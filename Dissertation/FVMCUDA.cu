#include "FVMCUDA.cuh"

/*__host__ void PhysicsCUDA::AllocateSparseMatrixMemoryToCPU() {
    SparseMatrix = new double[COOFORMAT];
    SparseColumnIndex = new int[COOFORMAT];
    SparseRowIndex = new int[COOFORMAT];
}

__host__ void PhysicsCUDA::DeAllocateSparseMatrixMemoryToCPU() {
    delete[] SparseMatrix;
    delete[] SparseColumnIndex;
    delete[] SparseRowIndex;
}*/

__host__ void PhysicsCUDA::CudaErrorChecker(hipError_t func) {
    if (debug) {
        if (func == hipSuccess) { std::cout << " CudaSuccess" << std::endl; }
        else { std::cout << stderr << " cudaFailure: " << hipGetErrorString(func) << std::endl; }
    }
    else { 
        if (func == hipSuccess) { ; }
        else { std::cout << stderr << " cudaFailure: " << hipGetErrorString(func) << std::endl; }
    }
}

__host__ void PhysicsCUDA::AllocateVariablesMemoryToGPU() {
    CudaErrorChecker(hipMalloc(&DeviceVars, VARALLOCSIZE));
}

__host__ void PhysicsCUDA::AllocateMatrixMemoryToGPU() {
    CudaErrorChecker(hipMalloc(&DeviceMatrix, VEC3ALLOCSIZE));
}

__host__ void PhysicsCUDA::AllocateInterimMemoryToGPU() {
    CudaErrorChecker(hipMalloc(&DeviceInterim, VEC2ALLOCSIZE));
}

__host__ void PhysicsCUDA::AllocateLinearSystemMemoryToGPU() {
    CudaErrorChecker(hipMalloc(&DeviceRHSVector, DOUBLEALLOCSIZE));
    CudaErrorChecker(hipMalloc(&DeviceNzCoeffMat, DOUBLESPARSEALLOCSIZE));
    CudaErrorChecker(hipMalloc(&DeviceSparseIndexI, INTALLOCSIZE));
    CudaErrorChecker(hipMalloc(&DeviceSparseIndexJ, INTALLOCSIZE));
}

__host__ void PhysicsCUDA::SetVariablesToGPU() {
    CudaErrorChecker(hipMemcpy(DeviceVars, GetVariableList(), VARALLOCSIZE, hipMemcpyHostToDevice));
}

__host__ void PhysicsCUDA::SetMatrixToGPU() {
    CudaErrorChecker(hipMemcpy(DeviceMatrix, GetSystemMatrix(), VEC3ALLOCSIZE, hipMemcpyHostToDevice));
}

__host__ void PhysicsCUDA::SetInterimToGPU() {
    CudaErrorChecker(hipMemcpy(DeviceInterim, GetInterimMatrix(), VEC2ALLOCSIZE, hipMemcpyHostToDevice));
}

__host__ void PhysicsCUDA::SetLinearSystemToGPU() {
    CudaErrorChecker(hipMemcpy(DeviceRHSVector, GetRHSVector(), DOUBLEALLOCSIZE, hipMemcpyHostToDevice));
    CudaErrorChecker(hipMemcpy(DeviceNzCoeffMat, GetnzCoeffMat(), DOUBLESPARSEALLOCSIZE, hipMemcpyHostToDevice));
    CudaErrorChecker(hipMemcpy(DeviceSparseIndexI, GetSparseIndexI(), INTALLOCSIZE, hipMemcpyHostToDevice));
    CudaErrorChecker(hipMemcpy(DeviceSparseIndexJ, GetSparseIndexJ(), INTALLOCSIZE, hipMemcpyHostToDevice));
}

__host__ void PhysicsCUDA::GetMatrixFromGPU() {
    CudaErrorChecker(hipMemcpy(GetSystemMatrix(), DeviceMatrix, VEC3ALLOCSIZE, hipMemcpyDeviceToHost));
}

__host__ void PhysicsCUDA::GetInterimFromGPU() {
    CudaErrorChecker(hipMemcpy(GetInterimMatrix(), DeviceInterim, VEC2ALLOCSIZE, hipMemcpyDeviceToHost));
}

__host__ void PhysicsCUDA::GetLinearSystemFromGPU() {
    CudaErrorChecker(hipMemcpy(GetRHSVector(), DeviceRHSVector, DOUBLEALLOCSIZE, hipMemcpyDeviceToHost));
    CudaErrorChecker(hipMemcpy(GetnzCoeffMat(), DeviceNzCoeffMat, DOUBLESPARSEALLOCSIZE, hipMemcpyDeviceToHost));
    CudaErrorChecker(hipMemcpy(GetSparseIndexI(), DeviceSparseIndexI, INTALLOCSIZE, hipMemcpyDeviceToHost));
    CudaErrorChecker(hipMemcpy(GetSparseIndexJ(), DeviceSparseIndexJ, INTALLOCSIZE, hipMemcpyDeviceToHost));
}

__host__ void PhysicsCUDA::DeviceVariablesCleanUp() { hipFree(DeviceVars); }

__host__ void PhysicsCUDA::DeviceMatrixCleanUp() { hipFree(DeviceMatrix); }

__host__ void PhysicsCUDA::DeviceInterimCleanUp() { hipFree(DeviceInterim); }

__host__ void PhysicsCUDA::DeviceLinearSystemCleanUp() { 
    hipFree(DeviceNzCoeffMat); 
    hipFree(DeviceSparseIndexI); 
    hipFree(DeviceSparseIndexJ); 
}

/*__host__ void PhysicsCUDA::DeviceSparseMatrixCleanUp() {
    hipFree(DeviceSparseMatrix);
    hipFree(DeviceColumnIndex);
    hipFree(DeviceRowIndex);
}*/

__device__ vec2 PhysicsCUDA::GetSPLITS(double* DeviceVars) { return vec2(DeviceVars[0], DeviceVars[1]); }

__device__ vec2 PhysicsCUDA::GetD(double* DeviceVars) { return vec2(DeviceVars[2], DeviceVars[3]); }

__device__ double PhysicsCUDA::GetDT(double* DeviceVars) { return DeviceVars[4]; }

__device__ double PhysicsCUDA::GetNU(double* DeviceVars) { return DeviceVars[5]; }

__device__ vec4 PhysicsCUDA::GetVelocityBoundary(double* DeviceVars) { return vec4(DeviceVars[6], DeviceVars[7], DeviceVars[8], DeviceVars[9]); }

__device__ vec3 PhysicsCUDA::GetMatrixValue(int i, int j, double* DeviceVars, vec3* DeviceMatrix) {
    int index = (j * (int)GetSPLITS(DeviceVars).y) + i;
    return DeviceMatrix[index];
}

__device__ vec2 PhysicsCUDA::GetInterimValue(int i, int j, double* DeviceVars, vec2* DeviceInterim) {
    int index = (j * (int)GetSPLITS(DeviceVars).y) + i;
    return DeviceInterim[index];
}

__device__ void PhysicsCUDA::SetMatrixValue(int i, int j, double var, int dim, double* DeviceVars, vec3* DeviceMatrix) {
    int index = (j * (int)GetSPLITS(DeviceVars).y) + i;
    if (dim == 0) { DeviceMatrix[index].u = var; }
    else if (dim == 1) { DeviceMatrix[index].v = var; }
    else if (dim == 2) { DeviceMatrix[index].p = var; }
}

__device__ void PhysicsCUDA::SetInterimValue(int i, int j, double var, int dim, double* DeviceVars, vec2* DeviceInterim) {
    int index = (j * (int)GetSPLITS(DeviceVars).y) + i;
    if (dim == 0) { DeviceInterim[index].x = var; }
    else if (dim == 1) { DeviceInterim[index].y = var; }
}

__device__ void PhysicsCUDA::SetLinearValue(int i, int j, double var, int dim, double* DeviceVars, double* DeviceRHSVector, double* DeviceNzCoeffMat, int* DeviceSparseIndexI, int* DeviceSparseIndexJ) {
    int TRUEindex = (j * (int)GetSPLITS(DeviceVars).y) + i;
    if (dim == 0) { DeviceRHSVector[TRUEindex] = var; }
    if (dim == 1) {
        DeviceSparseIndexI[TRUEindex] = TRUEindex;
        DeviceSparseIndexJ[TRUEindex] = TRUEindex;
        DeviceNzCoeffMat[TRUEindex] = var;
    }
    if (dim == 2) { 
        int index = (j * GetSPLITS(DeviceVars).y) + i + (GetSPLITS(DeviceVars).y * GetSPLITS(DeviceVars).x);
        DeviceSparseIndexI[index] = TRUEindex + 1;
        DeviceSparseIndexJ[index] = TRUEindex;
        DeviceNzCoeffMat[index] = var;
    }
    if (dim == 3) {
        int index = (j * GetSPLITS(DeviceVars).y) + i + (2 * (GetSPLITS(DeviceVars).y * GetSPLITS(DeviceVars).x)); 
        DeviceSparseIndexI[index] = TRUEindex - 1;
        DeviceSparseIndexJ[index] = TRUEindex;
        DeviceNzCoeffMat[index] = var;  
    }
    if (dim == 4) {
        int index = (j * GetSPLITS(DeviceVars).y) + i + (3 * (GetSPLITS(DeviceVars).y * GetSPLITS(DeviceVars).x)); 
        DeviceSparseIndexI[index] = TRUEindex;
        DeviceSparseIndexJ[index] = TRUEindex - GetSPLITS(DeviceVars).y;
        DeviceNzCoeffMat[index] = var; 
    }
    if (dim == 5) {
        int index = (j * GetSPLITS(DeviceVars).y) + i + (4 * (GetSPLITS(DeviceVars).y * GetSPLITS(DeviceVars).x));
        DeviceSparseIndexI[index] = TRUEindex;
        DeviceSparseIndexJ[index] = TRUEindex + GetSPLITS(DeviceVars).y;
        DeviceNzCoeffMat[index] = var;  
    }
}

__device__ vec6 PhysicsCUDA::InterpolateVelocities(int i, int j, int dim, double* DeviceVars, vec3* DeviceMatrix) {
    if (dim == 0) {
        double UEAST = 0.5 * (GetMatrixValue(i + 1, j, DeviceVars, DeviceMatrix).u + GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u);
        double UWEST = 0.5 * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u + GetMatrixValue(i - 1, j, DeviceVars, DeviceMatrix).u);
        double VNORTH = 0.5 * (GetMatrixValue(i - 1, j + 1, DeviceVars, DeviceMatrix).v + GetMatrixValue(i, j + 1, DeviceVars, DeviceMatrix).v);
        double VSOUTH = 0.5 * (GetMatrixValue(i - 1, j, DeviceVars, DeviceMatrix).v + GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v);

        if (j == 0) {
            double UNORTH = 0.5 * (GetMatrixValue(i, j + 1, DeviceVars, DeviceMatrix).u + GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u);
            double USOUTH = GetVelocityBoundary(DeviceVars).E;
            return vec6(UEAST, UWEST, UNORTH, USOUTH, VNORTH, VSOUTH);
        }
        else if (j == GetSPLITS(DeviceVars).y - 1) {
            double UNORTH = GetVelocityBoundary(DeviceVars).W;
            double USOUTH = 0.5 * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u + GetMatrixValue(i, j - 1, DeviceVars, DeviceMatrix).u);
            return vec6(UEAST, UWEST, UNORTH, USOUTH, VNORTH, VSOUTH);
        }
        else {
            double UNORTH = 0.5 * (GetMatrixValue(i, j + 1, DeviceVars, DeviceMatrix).u + GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u);
            double USOUTH = 0.5 * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u + GetMatrixValue(i, j - 1, DeviceVars, DeviceMatrix).u);
            return vec6(UEAST, UWEST, UNORTH, USOUTH, VNORTH, VSOUTH);
        }
    }
    else if (dim == 1) {
        double VNORTH = 0.5 * (GetMatrixValue(i, j + 1, DeviceVars, DeviceMatrix).v + GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v);
        double VSOUTH = 0.5 * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v + GetMatrixValue(i, j - 1, DeviceVars, DeviceMatrix).v);
        double UEAST = 0.5 * (GetMatrixValue(i + 1, j - 1, DeviceVars, DeviceMatrix).u + GetMatrixValue(i + 1, j, DeviceVars, DeviceMatrix).u);
        double UWEST = 0.5 * (GetMatrixValue(i, j - 1, DeviceVars, DeviceMatrix).u + GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u);

        if (i == 0) {
            double VEAST = 0.5 * (GetMatrixValue(i + 1, j, DeviceVars, DeviceMatrix).v + GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v);
            double VWEST = GetVelocityBoundary(DeviceVars).S;
            return vec6(VEAST, VWEST, VNORTH, VSOUTH, UEAST, UWEST);
        }
        else if (i == GetSPLITS(DeviceVars).x - 1) {
            double VEAST = GetVelocityBoundary(DeviceVars).N;
            double VWEST = 0.5 * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v + GetMatrixValue(i - 1, j, DeviceVars, DeviceMatrix).v);
            return vec6(VEAST, VWEST, VNORTH, VSOUTH, UEAST, UWEST);
        }
        else {
            double VEAST = 0.5 * (GetMatrixValue(i + 1, j, DeviceVars, DeviceMatrix).v + GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v);
            double VWEST = 0.5 * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v + GetMatrixValue(i - 1, j, DeviceVars, DeviceMatrix).v);
            return vec6(VEAST, VWEST, VNORTH, VSOUTH, UEAST, UWEST);
        }
    }
    //    else {
    //        std::cout << "No dimension found" << std::endl;
    //        throw - 1;
    //    }
}

__device__ double PhysicsCUDA::ComputeAdvection(int i, int j, int dim, double* DeviceVars, vec3* DeviceMatrix) {
    if (dim == 0) {
        vec6 var1 = InterpolateVelocities(i, j, dim, DeviceVars, DeviceMatrix);
        double XX = (var1.E * var1.E - var1.W * var1.W) / GetD(DeviceVars).x;
        double XY = (var1.N * var1.EN - var1.S * var1.WS) / GetD(DeviceVars).y;
        return -(XX + XY);
    }

    else if (dim == 1) {
        vec6 var2 = InterpolateVelocities(i, j, dim, DeviceVars, DeviceMatrix);
        double YY = (var2.N * var2.N - var2.S * var2.S) / GetD(DeviceVars).y;
        double YX = (var2.E * var2.EN - var2.W * var2.WS) / GetD(DeviceVars).x;
        return -(YY + YX);
    }
}

__device__ double PhysicsCUDA::ComputeDiffusion(int i, int j, int dim, double* DeviceVars, vec3* DeviceMatrix) {
    if (dim == 0) {
        double XDXe = -2 * GetNU(DeviceVars) * (GetMatrixValue(i + 1, j, DeviceVars, DeviceMatrix).u - GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u) / GetD(DeviceVars).x;
        double XDXw = -2 * GetNU(DeviceVars) * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u - GetMatrixValue(i - 1, j, DeviceVars, DeviceMatrix).u) / GetD(DeviceVars).x;

        if (j == GetSPLITS(DeviceVars).y - 1) {
            double XDYn = -GetNU(DeviceVars) * (GetVelocityBoundary(DeviceVars).E - GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u) / (GetD(DeviceVars).y / 2) -
                           GetNU(DeviceVars) * (GetMatrixValue(i, j + 1, DeviceVars, DeviceMatrix).v - GetMatrixValue(i - 1, j + 1, DeviceVars, DeviceMatrix).v) / GetD(DeviceVars).x;
            double XDYs = -GetNU(DeviceVars) * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u - GetMatrixValue(i, j - 1, DeviceVars, DeviceMatrix).u) / GetD(DeviceVars).y -
                           GetNU(DeviceVars) * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v - GetMatrixValue(i - 1, j, DeviceVars, DeviceMatrix).v) / GetD(DeviceVars).x;
            return (XDXe - XDXw) / GetD(DeviceVars).x + (XDYn - XDYs) / GetD(DeviceVars).y;
        }
        if (j == 0) {
            double XDYn = -GetNU(DeviceVars) * (GetMatrixValue(i, j + 1, DeviceVars, DeviceMatrix).u - GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u) / GetD(DeviceVars).y -
                           GetNU(DeviceVars) * (GetMatrixValue(i, j + 1, DeviceVars, DeviceMatrix).v - GetMatrixValue(i - 1, j + 1, DeviceVars, DeviceMatrix).v) / GetD(DeviceVars).x;
            double XDYs = -GetNU(DeviceVars) * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u - GetVelocityBoundary(DeviceVars).W) / (GetD(DeviceVars).y / 2) -
                           GetNU(DeviceVars) * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v - GetMatrixValue(i - 1, j, DeviceVars, DeviceMatrix).v) / GetD(DeviceVars).x;
            return (XDXe - XDXw) / GetD(DeviceVars).x + (XDYn - XDYs) / GetD(DeviceVars).y;
        }
        else {
            double XDYn = -GetNU(DeviceVars) * (GetMatrixValue(i, j + 1, DeviceVars, DeviceMatrix).u - GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u) / GetD(DeviceVars).y -
                           GetNU(DeviceVars) * (GetMatrixValue(i, j + 1, DeviceVars, DeviceMatrix).v - GetMatrixValue(i - 1, j + 1, DeviceVars, DeviceMatrix).v) / GetD(DeviceVars).x;
            double XDYs = -GetNU(DeviceVars) * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u - GetMatrixValue(i, j - 1, DeviceVars, DeviceMatrix).u) / GetD(DeviceVars).y -
                          GetNU(DeviceVars) * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v - GetMatrixValue(i - 1, j, DeviceVars, DeviceMatrix).v) / GetD(DeviceVars).x;
            return (XDXe - XDXw) / GetD(DeviceVars).x + (XDYn - XDYs) / GetD(DeviceVars).y;
        }
    }
    else if (dim == 1) {
        double YDYn = -2 * GetNU(DeviceVars) * (GetMatrixValue(i, j + 1, DeviceVars, DeviceMatrix).v - GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v) / GetD(DeviceVars).y;
        double YDYs = -2 * GetNU(DeviceVars) * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v - GetMatrixValue(i, j - 1, DeviceVars, DeviceMatrix).v) / GetD(DeviceVars).y;

        if (i == GetSPLITS(DeviceVars).x - 1) {
            double YDXe = -GetNU(DeviceVars) * (GetVelocityBoundary(DeviceVars).S - GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v) / (GetD(DeviceVars).x / 2) -
                           GetNU(DeviceVars) * (GetMatrixValue(i + 1, j, DeviceVars, DeviceMatrix).u - GetMatrixValue(i + 1, j - 1, DeviceVars, DeviceMatrix).u) / GetD(DeviceVars).y;
            double YDXw = -GetNU(DeviceVars) * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v - GetMatrixValue(i - 1, j, DeviceVars, DeviceMatrix).v) / GetD(DeviceVars).x -
                           GetNU(DeviceVars) * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u - GetMatrixValue(i, j - 1, DeviceVars, DeviceMatrix).u) / GetD(DeviceVars).y;
            return (YDYn - YDYs) / GetD(DeviceVars).y + (YDXe - YDXw) / GetD(DeviceVars).x;
        }
        if (i == 0) {
            double YDXe = -GetNU(DeviceVars) * (GetMatrixValue(i + 1, j, DeviceVars, DeviceMatrix).v - GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v) / GetD(DeviceVars).x -
                           GetNU(DeviceVars) * (GetMatrixValue(i + 1, j, DeviceVars, DeviceMatrix).u - GetMatrixValue(i + 1, j - 1, DeviceVars, DeviceMatrix).u) / GetD(DeviceVars).y;
            double YDXw = -GetNU(DeviceVars) * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v - GetVelocityBoundary(DeviceVars).N) / (GetD(DeviceVars).x / 2) -
                           GetNU(DeviceVars) * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u - GetMatrixValue(i, j - 1, DeviceVars, DeviceMatrix).u) / GetD(DeviceVars).y;
            return (YDYn - YDYs) / GetD(DeviceVars).y + (YDXe - YDXw) / GetD(DeviceVars).x;
        }
        else {
            double YDXe = -GetNU(DeviceVars) * (GetMatrixValue(i + 1, j, DeviceVars, DeviceMatrix).v - GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v) / GetD(DeviceVars).x -
                           GetNU(DeviceVars) * (GetMatrixValue(i + 1, j, DeviceVars, DeviceMatrix).u - GetMatrixValue(i + 1, j - 1, DeviceVars, DeviceMatrix).u) / GetD(DeviceVars).y;
            double YDXw = -GetNU(DeviceVars) * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v - GetMatrixValue(i - 1, j, DeviceVars, DeviceMatrix).v) / GetD(DeviceVars).x -
                           GetNU(DeviceVars) * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u - GetMatrixValue(i, j - 1, DeviceVars, DeviceMatrix).u) / GetD(DeviceVars).y;
            return (YDYn - YDYs) / GetD(DeviceVars).y + (YDXe - YDXw) / GetD(DeviceVars).x;
        }
    }
}

__device__ double PhysicsCUDA::ComputeMomentum(int i, int j, int dim, double* DeviceVars, vec3* DeviceMatrix) {
    double var = (ComputeAdvection(i, j, dim, DeviceVars, DeviceMatrix) - ComputeDiffusion(i, j, dim, DeviceVars, DeviceMatrix));
    return var;
}

__device__ void PhysicsCUDA::SetBaseAValues(int i, int j, double* DeviceVars, double* DeviceRHSVector, double* DeviceNzCoeffMat, int* DeviceSparseIndexI, int* DeviceSparseIndexJ) {
    double var = -GetDT(DeviceVars) * (GetD(DeviceVars).y / GetD(DeviceVars).x);
    if (i < GetSPLITS(DeviceVars).x && j < GetSPLITS(DeviceVars).y) {
        SetLinearValue(i, j, var, 2, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
        SetLinearValue(i, j, var, 3, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
        SetLinearValue(i, j, var, 4, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
        SetLinearValue(i, j, var, 5, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
    }
}

__device__ void PhysicsCUDA::BuildTopLeft(int i, int j, double* DeviceVars, vec3* DeviceMatrix, vec2* DeviceInterim, double* DeviceRHSVector, double* DeviceNzCoeffMat, int* DeviceSparseIndexI, int* DeviceSparseIndexJ) {
    if (i == 0 && j == 0) {
        double var;
        var = -GetD(DeviceVars).y * (GetMatrixValue(i + 1, j, DeviceVars, DeviceMatrix).u + GetDT(DeviceVars) * GetInterimValue(i + 1, j, DeviceVars, DeviceInterim).x) +
               GetD(DeviceVars).y * GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u -
               GetD(DeviceVars).x * (GetMatrixValue(i, j + 1, DeviceVars, DeviceMatrix).v + GetDT(DeviceVars) * GetInterimValue(i, j + 1, DeviceVars, DeviceInterim).y) +
               GetD(DeviceVars).x * GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v;
        SetLinearValue(i, j, var, 0, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
        var = (GetDT(DeviceVars) * (GetD(DeviceVars).y / GetD(DeviceVars).x)) + (GetDT(DeviceVars) * (GetD(DeviceVars).x / GetD(DeviceVars).y));
        SetLinearValue(i, j, var, 1, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
        SetLinearValue(i, j, 0.0, 3, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ); // No slip
        SetLinearValue(i, j, 0.0, 4, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ); // No slip
    }
}

__device__ void PhysicsCUDA::BuildTopRight(int i, int j, double* DeviceVars, vec3* DeviceMatrix, vec2* DeviceInterim, double* DeviceRHSVector, double* DeviceNzCoeffMat, int* DeviceSparseIndexI, int* DeviceSparseIndexJ) { // Bottom Left
    double var;
    if (i == 0 && j == (int)GetSPLITS(DeviceVars).y - 1) {
        var = -GetD(DeviceVars).y * (GetMatrixValue(i + 1, j, DeviceVars, DeviceMatrix).u + GetDT(DeviceVars) * GetInterimValue(i + 1, j, DeviceVars, DeviceInterim).x) +
               GetD(DeviceVars).y * GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u -
               GetD(DeviceVars).x * GetMatrixValue(i, j + 1, DeviceVars, DeviceMatrix).v +
               GetD(DeviceVars).x * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v + GetDT(DeviceVars) * GetInterimValue(i, j, DeviceVars, DeviceInterim).y);
        SetLinearValue(i, j, var, 0, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
        var = (GetDT(DeviceVars) * (GetD(DeviceVars).y / GetD(DeviceVars).x)) + GetDT(DeviceVars) * (GetD(DeviceVars).x / GetD(DeviceVars).y);
        SetLinearValue(i, j, var, 1, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
        SetLinearValue(i, j, 0.0, 3, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
        SetLinearValue(i, j, 0.0, 5, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
    }
}

__device__ void PhysicsCUDA::BuildBottomLeft(int i, int j, double* DeviceVars, vec3* DeviceMatrix, vec2* DeviceInterim, double* DeviceRHSVector, double* DeviceNzCoeffMat, int* DeviceSparseIndexI, int* DeviceSparseIndexJ) { // Top Right
    double var;
    if (i == (int)GetSPLITS(DeviceVars).x - 1 && j == 0) {
        var = -GetD(DeviceVars).y * GetMatrixValue(i + 1, j, DeviceVars, DeviceMatrix).u +
               GetD(DeviceVars).y * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u + GetDT(DeviceVars) * GetInterimValue(i, j, DeviceVars, DeviceInterim).x) -
               GetD(DeviceVars).x * (GetMatrixValue(i, j + 1, DeviceVars, DeviceMatrix).v + GetDT(DeviceVars) * GetInterimValue(i, j + 1, DeviceVars, DeviceInterim).y) +
               GetD(DeviceVars).x * GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v;
        SetLinearValue(i, j, var, 0, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
        var = (GetDT(DeviceVars) * (GetD(DeviceVars).y / GetD(DeviceVars).x)) + (GetDT(DeviceVars) * (GetD(DeviceVars).x / GetD(DeviceVars).y));
        SetLinearValue(i, j, var, 1, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
        SetLinearValue(i, j, 0.0, 2, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
        SetLinearValue(i, j, 0.0, 4, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
    }
}

__device__ void PhysicsCUDA::BuildBottomRight(int i, int j, double* DeviceVars, vec3* DeviceMatrix, vec2* DeviceInterim, double* DeviceRHSVector, double* DeviceNzCoeffMat, int* DeviceSparseIndexI, int* DeviceSparseIndexJ) {
    double var;
    if (i == (int)GetSPLITS(DeviceVars).x - 1 && j == (int)GetSPLITS(DeviceVars).y - 1) {
        var = -GetD(DeviceVars).y * GetMatrixValue(i + 1, j, DeviceVars, DeviceMatrix).u +
               GetD(DeviceVars).y * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u + GetDT(DeviceVars) * GetInterimValue(i, j, DeviceVars, DeviceInterim).x) -
               GetD(DeviceVars).x * GetMatrixValue(i, j + 1, DeviceVars, DeviceMatrix).v +
               GetD(DeviceVars).x * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v + GetDT(DeviceVars) * GetInterimValue(i, j, DeviceVars, DeviceInterim).y);
        SetLinearValue(i, j, var, 0, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
        var = (GetDT(DeviceVars) * (GetD(DeviceVars).y / GetD(DeviceVars).x)) + (GetDT(DeviceVars) * (GetD(DeviceVars).x / GetD(DeviceVars).y));
        SetLinearValue(i, j, var, 1, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
        SetLinearValue(i, j, 0.0, 2, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
        SetLinearValue(i, j, 0.0, 5, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
    }
}

__device__ void PhysicsCUDA::BuildLeftSide(int i, int j, double* DeviceVars, vec3* DeviceMatrix, vec2* DeviceInterim, double* DeviceRHSVector, double* DeviceNzCoeffMat, int* DeviceSparseIndexI, int* DeviceSparseIndexJ) { // Top side
    double var;
    if (j == 0) {
        if (i > 0 && i < GetSPLITS(DeviceVars).x - 1) {
            var = -GetD(DeviceVars).y * (GetMatrixValue(i + 1, j, DeviceVars, DeviceMatrix).u + GetDT(DeviceVars) * GetInterimValue(i + 1, j, DeviceVars, DeviceInterim).x) +
                   GetD(DeviceVars).y * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u + GetDT(DeviceVars) * GetInterimValue(i, j, DeviceVars, DeviceInterim).x) -
                   GetD(DeviceVars).x * (GetMatrixValue(i, j + 1, DeviceVars, DeviceMatrix).v + GetDT(DeviceVars) * GetInterimValue(i, j + 1, DeviceVars, DeviceInterim).y) +
                   GetD(DeviceVars).x * GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v;
            SetLinearValue(i, j, var, 0, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
            var = (2 * GetDT(DeviceVars) * (GetD(DeviceVars).y / GetD(DeviceVars).x)) + (GetDT(DeviceVars) * (GetD(DeviceVars).x / GetD(DeviceVars).y));
            SetLinearValue(i, j, var, 1, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
            SetLinearValue(i, j, 0.0, 4, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
        }
    }
}

__device__ void PhysicsCUDA::BuildRightSide(int i, int j, double* DeviceVars, vec3* DeviceMatrix, vec2* DeviceInterim, double* DeviceRHSVector, double* DeviceNzCoeffMat, int* DeviceSparseIndexI, int* DeviceSparseIndexJ) {
    double var;
    if (j == GetSPLITS(DeviceVars).y - 1) {
        if (i > 0 && i < GetSPLITS(DeviceVars).x - 1) {
            var = -GetD(DeviceVars).y * (GetMatrixValue(i + 1, j, DeviceVars, DeviceMatrix).u + GetDT(DeviceVars) * GetInterimValue(i + 1, j, DeviceVars, DeviceInterim).x) +
                   GetD(DeviceVars).y * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u + GetDT(DeviceVars) * GetInterimValue(i, j, DeviceVars, DeviceInterim).x) -
                   GetD(DeviceVars).x * GetMatrixValue(i, j + 1, DeviceVars, DeviceMatrix).v +
                   GetD(DeviceVars).x * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v + GetDT(DeviceVars) * GetInterimValue(i, j, DeviceVars, DeviceInterim).y);
            SetLinearValue(i, j, var, 0, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
            var = (2 * GetDT(DeviceVars) * (GetD(DeviceVars).y / GetD(DeviceVars).x)) + (GetDT(DeviceVars) * (GetD(DeviceVars).x / GetD(DeviceVars).y));
            SetLinearValue(i, j, var, 1, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
            SetLinearValue(i, j, 0.0, 5, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
        }
    }
}

__device__ void PhysicsCUDA::BuildTopSide(int i, int j, double* DeviceVars, vec3* DeviceMatrix, vec2* DeviceInterim, double* DeviceRHSVector, double* DeviceNzCoeffMat, int* DeviceSparseIndexI, int* DeviceSparseIndexJ) {
    double var;
    if (i == 0) {
        if (j > 0 && j < GetSPLITS(DeviceVars).y - 1) {
            var = -GetD(DeviceVars).y * (GetMatrixValue(i + 1, j, DeviceVars, DeviceMatrix).u + GetDT(DeviceVars) * GetInterimValue(i + 1, j, DeviceVars, DeviceInterim).x) +
                   GetD(DeviceVars).y * GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u -
                   GetD(DeviceVars).x * (GetMatrixValue(i, j + 1, DeviceVars, DeviceMatrix).v + GetDT(DeviceVars) * GetInterimValue(i, j + 1, DeviceVars, DeviceInterim).y) +
                   GetD(DeviceVars).x * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v + GetDT(DeviceVars) * GetInterimValue(i, j, DeviceVars, DeviceInterim).y);
            SetLinearValue(i, j, var, 0, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
            var = (GetDT(DeviceVars) * (GetD(DeviceVars).y / GetD(DeviceVars).x)) + (2 * GetDT(DeviceVars) * (GetD(DeviceVars).x / GetD(DeviceVars).y));
            SetLinearValue(i, j, var, 1, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
            SetLinearValue(i, j, 0.0, 3, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
        }
    }
}

__device__ void PhysicsCUDA::BuildBottomSide(int i, int j, double* DeviceVars, vec3* DeviceMatrix, vec2* DeviceInterim, double* DeviceRHSVector, double* DeviceNzCoeffMat, int* DeviceSparseIndexI, int* DeviceSparseIndexJ) {
    double var;
    if (i == GetSPLITS(DeviceVars).x - 1) {
        if (j > 0 && j < GetSPLITS(DeviceVars).y - 1) {
            var = -GetD(DeviceVars).y * GetMatrixValue(i + 1, j, DeviceVars, DeviceMatrix).u +
                   GetD(DeviceVars).y * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u + GetDT(DeviceVars) * GetInterimValue(i, j, DeviceVars, DeviceInterim).x) -
                   GetD(DeviceVars).x * (GetMatrixValue(i, j + 1, DeviceVars, DeviceMatrix).v + GetDT(DeviceVars) * GetInterimValue(i, j + 1, DeviceVars, DeviceInterim).y) +
                   GetD(DeviceVars).x * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v + GetDT(DeviceVars) * GetInterimValue(i, j, DeviceVars, DeviceInterim).y);
            SetLinearValue(i, j, var, 0, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
            var = (GetDT(DeviceVars) * (GetD(DeviceVars).y / GetD(DeviceVars).x)) + (2 * GetDT(DeviceVars) * (GetD(DeviceVars).x / GetD(DeviceVars).y));
            SetLinearValue(i, j, var, 1, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
            SetLinearValue(i, j, 0.0, 2, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
        }
    }
}

__device__ void PhysicsCUDA::BuildInterior(int i, int j, double* DeviceVars, vec3* DeviceMatrix, vec2* DeviceInterim, double* DeviceRHSVector, double* DeviceNzCoeffMat, int* DeviceSparseIndexI, int* DeviceSparseIndexJ) {
    double var;
    if (i > 0 && i < GetSPLITS(DeviceVars).x - 1) {
        if (j > 0 && j < GetSPLITS(DeviceVars).y - 1) {
            var = -GetD(DeviceVars).y * (GetMatrixValue(i + 1, j, DeviceVars, DeviceMatrix).u + GetDT(DeviceVars) * GetInterimValue(i + 1, j, DeviceVars, DeviceInterim).x) +
                   GetD(DeviceVars).y * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u + GetDT(DeviceVars) * GetInterimValue(i, j, DeviceVars, DeviceInterim).x) -
                   GetD(DeviceVars).x * (GetMatrixValue(i, j + 1, DeviceVars, DeviceMatrix).v + GetDT(DeviceVars) * GetInterimValue(i, j + 1, DeviceVars, DeviceInterim).y) +
                   GetD(DeviceVars).x * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v + GetDT(DeviceVars) * GetInterimValue(i, j, DeviceVars, DeviceInterim).y);
            SetLinearValue(i, j, var, 0, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
            var = (2 * GetDT(DeviceVars) * (GetD(DeviceVars).y / GetD(DeviceVars).x)) + (2 * GetDT(DeviceVars) * (GetD(DeviceVars).x / GetD(DeviceVars).y));
            SetLinearValue(i, j, var, 1, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
        }
    }
}

__device__ void PhysicsCUDA::BuildLinearSystem(int i, int j, double* DeviceVars, vec3* DeviceMatrix, vec2* DeviceInterim, double* DeviceRHSVector, double* DeviceNzCoeffMat, int* DeviceSparseIndexI, int* DeviceSparseIndexJ) {
    SetBaseAValues(i, j, DeviceVars, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
    BuildTopLeft(i, j, DeviceVars, DeviceMatrix, DeviceInterim, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
    BuildTopRight(i, j, DeviceVars, DeviceMatrix, DeviceInterim, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
    BuildBottomLeft(i, j, DeviceVars, DeviceMatrix, DeviceInterim, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
    BuildBottomRight(i, j, DeviceVars, DeviceMatrix, DeviceInterim, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
    BuildLeftSide(i, j, DeviceVars, DeviceMatrix, DeviceInterim, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
    BuildRightSide(i, j, DeviceVars, DeviceMatrix, DeviceInterim, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
    BuildTopSide(i, j, DeviceVars, DeviceMatrix, DeviceInterim, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
    BuildBottomSide(i, j, DeviceVars, DeviceMatrix, DeviceInterim, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
    BuildInterior(i, j, DeviceVars, DeviceMatrix, DeviceInterim, DeviceRHSVector, DeviceNzCoeffMat, DeviceSparseIndexI, DeviceSparseIndexJ);
    return;
}

//__device__ void PhysicsCUDA::BuildSparseMatrixForSolution(int i, int j, double* DeviceVars, double* DeviceRHSVector, double* DeviceNzCoeffMat, int* DeviceSparseIndexI, int* DeviceSparseIndexJ) {
//    int index = (j * GetSPLITS(DeviceVars).x * GetSPLITS(DeviceVars).y) + i;
//    int nnz = 5 * GetSPLITS(DeviceVars).x * GetSPLITS(DeviceVars).y;
//    if (index < nnz) {
//        //printf("I = %i, J = %i, Value = %f\n", DeviceSparseIndexI[index], DeviceSparseIndexJ[index], DeviceNzCoeffMat[index]);
//    }
//}

__device__ double PhysicsCUDA::ComputeIteration(int i, int j, int dim, double* DeviceVars, vec3* DeviceMatrix, vec2* DeviceInterim) {
    if (dim == 0) {
        double var = GetMatrixValue(i, j, DeviceVars, DeviceMatrix).u + (GetDT(DeviceVars) * GetInterimValue(i, j, DeviceVars, DeviceInterim).x) -
                     GetDT(DeviceVars) * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).p - GetMatrixValue(i - 1, j, DeviceVars, DeviceMatrix).p) / GetD(DeviceVars).x;
        return var;
    }
    else if (dim == 1) {
        double var = GetMatrixValue(i, j, DeviceVars, DeviceMatrix).v + (GetDT(DeviceVars) * GetInterimValue(i, j, DeviceVars, DeviceInterim).y) -
                     GetDT(DeviceVars) * (GetMatrixValue(i, j, DeviceVars, DeviceMatrix).p - GetMatrixValue(i, j - 1, DeviceVars, DeviceMatrix).p) / GetD(DeviceVars).y;
        return var;
    }
}

/*void PhysicsCUDA::ThrowCoefficients() {
    std::ofstream CoeFile;
    CoeFile.open("./Output/Coefficients.txt");
    CoeFile << "| B | AC | AIP | AIN | AJP | AJN |" << std::endl;
    for (int i = 0; i < GetSPLITS().x * GetSPLITS().y; i++) {
        if (i < ((GetSPLITS().x * GetSPLITS().y) - GetSPLITS().x)) {
            CoeFile << GetLinearValue(i).Bvec << " , " << GetLinearValue(i).Acen << " , "
                    << GetLinearValue(i).Aipos << " , " << GetLinearValue(i + 1).Aisub << " , "
                    << GetLinearValue(i + (int)GetSPLITS().x).Ajpos << " , " << GetLinearValue(i).Ajsub << std::endl;
        }
        else {
            if (i < ((GetSPLITS().x * GetSPLITS().y) - 1)) {
                CoeFile << GetLinearValue(i).Bvec << " , " << GetLinearValue(i).Acen << " , "
                    << GetLinearValue(i).Aipos << " , " << GetLinearValue(i + 1).Aisub << std::endl;
            }
            else {
                CoeFile << GetLinearValue(i).Bvec << " , " << GetLinearValue(i).Acen << std::endl;
            }
        }
    }
    CoeFile.close();
    return;
}*/